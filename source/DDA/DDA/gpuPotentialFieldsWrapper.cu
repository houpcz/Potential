#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void countFields(float * devPotentialField, int fieldWidth, int tileWidth, float * fieldCenterX, float * fieldCenterY, int * goalX, int * goalY);

inline void
check_cuda_errors (const char *filename, const int line_number)
{
  hipDeviceSynchronize ();
  hipError_t error = hipGetLastError ();
  if (error != hipSuccess)
    {
      printf ("CUDA error at %s:%i: %s\n", filename, line_number,
	      hipGetErrorString (error));
      exit (-1);
    }
}


static void
HandleError (hipError_t error, const char *file, int line)
{
  if (error != hipSuccess)
    {
      printf ("%s in %s at line %d\n", hipGetErrorString (error), file,
	      line);
      exit (EXIT_FAILURE);
    }
}

#define CHECK_ERROR( error ) ( HandleError( error, __FILE__, __LINE__ ) )

float * cpuPotentialField;
float * devPotentialField;
float * devFieldCenterX;
float * devFieldCenterY;
int * devGoalX;
int * devGoalY;

int numberAgents;
int fieldWidth;
int tileWidth;

size_t memPotentialField;
size_t memGoal;
size_t memFieldCenter;

void gpuAllocMemory(int _numberAgents, int _fieldWidth, int _tileWidth)
{
	numberAgents = _numberAgents;
	fieldWidth = _fieldWidth;
	tileWidth = _tileWidth;

	memPotentialField = numberAgents * fieldWidth * fieldWidth * sizeof(float);
	memFieldCenter = numberAgents * sizeof(float);
	memGoal = numberAgents * sizeof(int);

	cpuPotentialField = (float *) malloc(memPotentialField);
	hipMalloc( (void**)&devPotentialField, memPotentialField);
	hipMalloc( (void**)&devFieldCenterX, memFieldCenter);
	hipMalloc( (void**)&devFieldCenterY, memFieldCenter);
	hipMalloc( (void**)&devGoalX, memGoal);
	hipMalloc( (void**)&devGoalY, memGoal);
}

void gpuCountPotentialFields(float *** potentialField, float * cpuFieldCenterX, float * cpuFieldCenterY, int * cpuGoalX, int * cpuGoalY)
{
	// grid configuration1
    dim3 gridRes(numberAgents, 1, 1);
    dim3 blockRes(fieldWidth / 4, fieldWidth / 4, 1);

	hipMemcpy( devFieldCenterX, cpuFieldCenterX, memFieldCenter, hipMemcpyHostToDevice ) ;
	hipMemcpy( devFieldCenterY, cpuFieldCenterY, memFieldCenter, hipMemcpyHostToDevice ) ;
	hipMemcpy( devGoalX, cpuGoalX, memGoal, hipMemcpyHostToDevice ) ;
	hipMemcpy( devGoalY, cpuGoalY, memGoal, hipMemcpyHostToDevice ) ;

    countFields<<< gridRes, blockRes >>>( devPotentialField, fieldWidth, tileWidth, devFieldCenterX, devFieldCenterY, devGoalX, devGoalY);
	CHECK_ERROR( hipGetLastError() );
	hipDeviceSynchronize();

	hipMemcpy( cpuPotentialField, devPotentialField, memPotentialField, hipMemcpyDeviceToHost ) ;

	for(int loop1 = 0; loop1 < numberAgents; loop1++)
		for(int loop2 = 0; loop2 < fieldWidth; loop2++)
			for(int loop3 = 0; loop3 < fieldWidth; loop3++)
				potentialField[loop1][loop2][loop3] = cpuPotentialField[loop1 * fieldWidth * fieldWidth + loop2 * fieldWidth + loop3];
}

void gpuFreeMemory()
{
	free(cpuPotentialField);
	hipFree(devPotentialField);
	hipFree(devFieldCenterX);
	hipFree(devFieldCenterY);
	hipFree(devGoalX);
	hipFree(devGoalY);
}