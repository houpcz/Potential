#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#include <math.h>
#include <algorithm>
#include "Triangle.h"

#define OBSTACLE 100000.0f
#define SMALL_OBSTACLE 10000.0f;
__global__ void countFields(float * devPotentialField, int fieldWidth, int tileWidth, float * fieldCenterX, float * fieldCenterY, int * goalX, int * goalY);
__global__ void smoothFields(float * devPotentialFieldIn, float *devPotentialFieldOut, int fieldWidth, int tileWidth);

inline void
check_cuda_errors (const char *filename, const int line_number)
{
  hipDeviceSynchronize ();
  hipError_t error = hipGetLastError ();
  if (error != hipSuccess)
    {
      printf ("CUDA error at %s:%i: %s\n", filename, line_number,
	      hipGetErrorString (error));
      exit (-1);
    }
}


static void
HandleError (hipError_t error, const char *file, int line)
{
  if (error != hipSuccess)
    {
      printf ("%s in %s at line %d\n", hipGetErrorString (error), file,
	      line);
      exit (EXIT_FAILURE);
    }
}

#define CHECK_ERROR( error ) ( HandleError( error, __FILE__, __LINE__ ) )

#define OBST_AREA_LEFT 0
#define OBST_AREA_TOP 1
#define CELL_WIDTH 2
#define CELL_HEIGHT 3
__constant__ int param[4];

__constant__ int quadTree[AREA_CELL_WIDTH * AREA_CELL_HEIGHT];
__constant__ Triangle triangle[MAX_TRIANGLE];
__constant__ int triangleIDs[MAX_TRIANGLE_IDS];

float * cpuPotentialField;
float * devPotentialFieldIn;
float * devPotentialFieldOut;
float * devFieldCenterX;
float * devFieldCenterY;
int * devGoalX;
int * devGoalY;

int numberAgents;
int fieldWidth;
int tileWidth;

size_t memPotentialField;
size_t memGoal;
size_t memFieldCenter;

void gpuAllocMemory(int _numberAgents, int _fieldWidth, int _tileWidth)
{
	numberAgents = _numberAgents;
	fieldWidth = _fieldWidth;
	tileWidth = _tileWidth;

	memPotentialField = numberAgents * fieldWidth * fieldWidth * sizeof(float);
	memFieldCenter = numberAgents * sizeof(float);
	memGoal = numberAgents * sizeof(int);

	cpuPotentialField = (float *) malloc(memPotentialField);
	hipMalloc( (void**)&devPotentialFieldIn, memPotentialField);
	hipMalloc( (void**)&devPotentialFieldOut, memPotentialField);
	hipMalloc( (void**)&devFieldCenterX, memFieldCenter);
	hipMalloc( (void**)&devFieldCenterY, memFieldCenter);
	hipMalloc( (void**)&devGoalX, memGoal);
	hipMalloc( (void**)&devGoalY, memGoal);
}

void gpuAllocObstacles(int _obstAreaLeft, int _obstAreaTop, int _cellWidth, int _cellHeight, int * _quadTree, Triangle *_triangle, int triangleSize, int * _triangleIDs, int triangleIDsSize)
{
	int _param[4];
	_param[0] = _obstAreaLeft;
	_param[1] = _obstAreaTop;
	_param[2] = _cellWidth;
	_param[3] = _cellHeight;

	hipMemcpyToSymbol((const char *) param, _param, sizeof(int) * 4) ;
	hipMemcpyToSymbol((const char *) triangle, _triangle, sizeof(Triangle) * triangleSize) ;
	hipMemcpyToSymbol((const char *) quadTree, _quadTree, sizeof(int) * AREA_CELL_WIDTH * AREA_CELL_HEIGHT) ;
	hipMemcpyToSymbol((const char *) triangleIDs, _triangleIDs, sizeof(int) * triangleIDsSize) ;
}

void gpuCountPotentialFields(float *** potentialField, float * cpuFieldCenterX, float * cpuFieldCenterY, int * cpuGoalX, int * cpuGoalY)
{
	// grid configuration1
    dim3 gridRes(numberAgents, 1, 1);
    dim3 blockRes(fieldWidth / 2, fieldWidth / 2, 1);

	hipMemcpy( devFieldCenterX, cpuFieldCenterX, memFieldCenter, hipMemcpyHostToDevice ) ;
	hipMemcpy( devFieldCenterY, cpuFieldCenterY, memFieldCenter, hipMemcpyHostToDevice ) ;
	hipMemcpy( devGoalX, cpuGoalX, memGoal, hipMemcpyHostToDevice ) ;
	hipMemcpy( devGoalY, cpuGoalY, memGoal, hipMemcpyHostToDevice ) ;

    countFields<<< gridRes, blockRes >>>( devPotentialFieldIn, fieldWidth, tileWidth, devFieldCenterX, devFieldCenterY, devGoalX, devGoalY);
	CHECK_ERROR( hipGetLastError() );
	hipDeviceSynchronize();
	smoothFields<<< gridRes, blockRes >>> (devPotentialFieldIn, devPotentialFieldOut, fieldWidth, tileWidth);
	hipMemcpy( cpuPotentialField, devPotentialFieldOut, memPotentialField, hipMemcpyDeviceToHost ) ;
	//hipMemcpy( cpuPotentialField, devPotentialFieldIn, memPotentialField, hipMemcpyDeviceToHost ) ;
	
	for(int loop1 = 0; loop1 < numberAgents; loop1++)
		for(int loop2 = 0; loop2 < fieldWidth; loop2++)
			for(int loop3 = 0; loop3 < fieldWidth; loop3++)
				potentialField[loop1][loop2][loop3] = cpuPotentialField[loop1 * fieldWidth * fieldWidth + loop2 * fieldWidth + loop3];
	
}

void gpuFreeMemory()
{
	free(cpuPotentialField);
	hipFree(devPotentialFieldIn);
	hipFree(devPotentialFieldOut);
	hipFree(devFieldCenterX);
	hipFree(devFieldCenterY);
	hipFree(devGoalX);
	hipFree(devGoalY);
}

#define MAX(a, b) (((a) > (b)) ? (a) : (b))

__device__ bool pointTriangleTest1(int x, int y, const Triangle & t)
{
	float ab = (t.p[0].x - x) * (t.p[1].y - y) - (t.p[1].x - x) * (t.p[0].y - y);
	float bc = (t.p[1].x - x) * (t.p[2].y - y) - (t.p[2].x - x) * (t.p[1].y - y);
	float ca = (t.p[2].x - x) * (t.p[0].y - y) - (t.p[0].x - x) * (t.p[2].y - y);

	return ((ab <= 0.0f && bc <= 0.0f && ca <= 0.0f) || (ab >= 0.0f && bc >= 0.0f && ca >= 0.0f));
}

__device__ bool pointTriangleTest2(int x, int y, const Triangle & t)
{
	float p0x = t.p[0].x - x;
	float p0y = t.p[0].y - y;
	float p1x = t.p[1].x - x;
	float p1y = t.p[1].y - y;
	float p2x = t.p[2].x - x;
	float p2y = t.p[2].y - y;
	float ab = p0x * p1y - p1x * p0y;
	float bc = p1x * p2y - p2x * p1y;
	float ca = p2x * p0y - p0x * p2y;

	return ((ab <= 0.0f && bc <= 0.0f && ca <= 0.0f) || (ab >= 0.0f && bc >= 0.0f && ca >= 0.0f));
}

__device__ bool pointTriangleTest3(int x, int y, const Triangle & t)
{
	float p0x = t.p[0].x - x;
	float p0y = t.p[0].y - y;
	float p1x = t.p[1].x - x;
	float p1y = t.p[1].y - y;
	if(p0x * p1y - p1x * p0y < 0.0f) // ab
		return false;
	float p2x = t.p[2].x - x;
	float p2y = t.p[2].y - y;
	float bc = p1x * p2y - p2x * p1y;
	float ca = p2x * p0y - p0x * p2y;

	return bc >= 0.0f && ca >= 0.0f;
}

__device__ bool pointTriangleTest(int x, int y, const Triangle & t)
{
	float p0x = t.p[0].x - x;
	float p0y = t.p[0].y - y;
	float p1x = t.p[1].x - x;
	float p1y = t.p[1].y - y;
	float p2x = t.p[2].x - x;
	float p2y = t.p[2].y - y;

	return (p0x * p1y - p1x * p0y) >= 0.0f && (p1x * p2y - p2x * p1y) >= 0.0f && (p2x * p0y - p0x * p2y) >= 0.0f;
}

__device__ float countFieldTile(int x, int y, int goalX, int goalY)
{
	float dx = x - goalX;
	float dy = y - goalY;
	float initValue = sqrtf(dx * dx + dy * dy);

	int areaX = x - param[OBST_AREA_LEFT];
	int areaY = y - param[OBST_AREA_TOP];
	float obst = 0.0f;

	if(areaX >= 0 && areaY >= 0 && 
	   areaX < param[CELL_WIDTH] * AREA_CELL_WIDTH &&
	   areaY < param[CELL_HEIGHT] * AREA_CELL_HEIGHT)
	{
		int yid = areaY / param[CELL_HEIGHT];
		int xid = areaX / param[CELL_WIDTH];
		int id = xid + yid * AREA_CELL_WIDTH;
		int startX;
		if(id == 0)
			startX = 0;
		else
			startX = quadTree[id - 1];
		int endX = quadTree[id];

		for(int loop1 = startX; loop1 < endX; loop1++)
		{
			int triangleID = triangleIDs[loop1];
			if(pointTriangleTest(x, y, triangle[triangleID]))
			{
				obst = OBSTACLE;
				break;
			}
		}
	}
	
	return initValue + obst;
}

__global__ void countFields(float * devPotentialField, int fieldWidth, int tileWidth, float * fieldCenterX, float * fieldCenterY, int * goalX, int * goalY)
{
	int fieldID = blockIdx.x;
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	int myWorkFieldWidth = fieldWidth / blockDim.x;
	int myWorkFieldHeight = fieldWidth / blockDim.y;
	int endWorkX = MAX((threadX + 1) * myWorkFieldWidth, fieldWidth);
	int endWorkY = MAX((threadY + 1) * myWorkFieldHeight, fieldWidth);
	int x, y;

	for(int loop1 = threadY * myWorkFieldWidth; loop1 < endWorkY; loop1++)
	{
		for(int loop2 = threadX * myWorkFieldWidth; loop2 < endWorkX; loop2++)
		{
			x = (int) fieldCenterX[fieldID] + (loop2 - (fieldWidth / 2)) * tileWidth; 
			y = (int) fieldCenterY[fieldID] + (loop1 - (fieldWidth / 2)) * tileWidth;
			devPotentialField[fieldID * fieldWidth * fieldWidth + loop1 * fieldWidth + loop2] = countFieldTile(x, y, goalX[fieldID], goalY[fieldID]);
		}
	}
}

__global__ void smoothFields(float * devPotentialFieldIn, float *devPotentialFieldOut, int fieldWidth, int tileWidth)
{
	int fieldID = blockIdx.x;
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	int myWorkFieldWidth = fieldWidth / blockDim.x;
	int myWorkFieldHeight = fieldWidth / blockDim.y;
	int endWorkX = MAX((threadX + 1) * myWorkFieldWidth, fieldWidth);
	int endWorkY = MAX((threadY + 1) * myWorkFieldHeight, fieldWidth);
	int arrayID;
	int fieldOffset = fieldID * fieldWidth * fieldWidth;
	for(int loop1 = threadY * myWorkFieldWidth; loop1 < endWorkY; loop1++)
	{
		for(int loop2 = threadX * myWorkFieldWidth; loop2 < endWorkX; loop2++)
		{
			arrayID = fieldOffset + loop1 * fieldWidth + loop2;
			if((loop2 > 0 && devPotentialFieldOut[arrayID - 1] > OBSTACLE) ||
			   (loop2 < fieldWidth - 1 && devPotentialFieldOut[arrayID + 1] > OBSTACLE) ||
			   (loop1 > 0 && devPotentialFieldOut[arrayID - fieldWidth] > OBSTACLE) ||
			   (loop1 < fieldWidth - 1 && devPotentialFieldOut[arrayID + fieldWidth] > OBSTACLE)) 
			{
				devPotentialFieldOut[arrayID] = devPotentialFieldIn[arrayID] + SMALL_OBSTACLE;
			} else 
			{
				devPotentialFieldOut[arrayID] = devPotentialFieldIn[arrayID];
			}
		}
	}
}