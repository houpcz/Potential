#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#include <hip/device_functions.h>
#include <math.h>

#define MAX(a, b) (((a) > (b)) ? (a) : (b))

/*
__device__ bool max(int val1, int val2)
{
	return (val1 > val2) ? val1 : val2;
}
*/

__device__ float countFieldTile(int x, int y, int goalX, int goalY)
{
	float dx = x - goalX;
	float dy = y - goalY;
	float initValue = sqrtf(dx * dx + dy * dy);

	/*
	bool inTriangle = false;
	for(int loop1 = 0; loop1 < obstacle->size(); loop1++)
	{
		if((*obstacle)[loop1]->polygon().containsPoint(QPointF(x, y), Qt::FillRule::WindingFill))
		{
			inTriangle = true;
			break;
		}
	}
	float obst = inTriangle ? PotentialField::OBSTACLE : 0;
	*/
	return initValue; // + obst;
}

__global__ void countFields(float * devPotentialField, int fieldWidth, int tileWidth, float * fieldCenterX, float * fieldCenterY, int * goalX, int * goalY)
{
	int fieldID = blockIdx.x;
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	int myWorkFieldWidth = fieldWidth / blockDim.x;
	int myWorkFieldHeight = fieldWidth / blockDim.y;
	int endWorkX = MAX((threadX + 1) * myWorkFieldWidth, fieldWidth);
	int endWorkY = MAX((threadY + 1) * myWorkFieldHeight, fieldWidth);
	int x, y;

	for(int loop1 = threadY * myWorkFieldWidth; loop1 < endWorkY; loop1++)
	{
		for(int loop2 = threadX * myWorkFieldWidth; loop2 < endWorkX; loop2++)
		{
			x = (int) fieldCenterX[fieldID] + (loop2 - (fieldWidth / 2)) * tileWidth; 
			y = (int) fieldCenterY[fieldID] + (loop1 - (fieldWidth / 2)) * tileWidth;
			devPotentialField[fieldID * fieldWidth * fieldWidth + loop1 * fieldWidth + loop2] = countFieldTile(x, y, goalX[fieldID], goalY[fieldID]);
		}
	}
}