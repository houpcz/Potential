#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <math.h>
#include "Triangle.h"

#define MAX(a, b) (((a) > (b)) ? (a) : (b))

__device__ bool pointTriangleTest(Point2d & p, Triangle & t)
{
	float ab = (t.p[0].x - p.x) * (t.p[1].y - p.y) - (t.p[1].x - p.x) * (t.p[0].y - p.y);
	float bc = (t.p[1].x - p.x) * (t.p[2].y - p.y) - (t.p[2].x - p.x) * (t.p[1].y - p.y);
	float ca = (t.p[2].x - p.x) * (t.p[0].y - p.y) - (t.p[0].x - p.x) * (t.p[2].y - p.y);
	
	return (ab <= 0.0f && bc <= 0.0f && ca <= 0.0f) || (ab >= 0.0f && bc >= 0.0f && ca >= 0.0f);
}

__device__ float countFieldTile(int x, int y, int goalX, int goalY)
{
	float dx = x - goalX;
	float dy = y - goalY;
	float initValue = sqrtf(dx * dx + dy * dy);

	/*
	bool inTriangle = false;
	for(int loop1 = 0; loop1 < obstacle->size(); loop1++)
	{
		if((*obstacle)[loop1]->polygon().containsPoint(QPointF(x, y), Qt::FillRule::WindingFill))
		{
			inTriangle = true;
			break;
		}
	}
	float obst = inTriangle ? PotentialField::OBSTACLE : 0;
	*/
	return initValue; // + obst;
}

__global__ void countFields(float * devPotentialField, int fieldWidth, int tileWidth, float * fieldCenterX, float * fieldCenterY, int * goalX, int * goalY)
{
	int fieldID = blockIdx.x;
	int threadX = threadIdx.x;
	int threadY = threadIdx.y;
	int myWorkFieldWidth = fieldWidth / blockDim.x;
	int myWorkFieldHeight = fieldWidth / blockDim.y;
	int endWorkX = MAX((threadX + 1) * myWorkFieldWidth, fieldWidth);
	int endWorkY = MAX((threadY + 1) * myWorkFieldHeight, fieldWidth);
	int x, y;

	for(int loop1 = threadY * myWorkFieldWidth; loop1 < endWorkY; loop1++)
	{
		for(int loop2 = threadX * myWorkFieldWidth; loop2 < endWorkX; loop2++)
		{
			x = (int) fieldCenterX[fieldID] + (loop2 - (fieldWidth / 2)) * tileWidth; 
			y = (int) fieldCenterY[fieldID] + (loop1 - (fieldWidth / 2)) * tileWidth;
			devPotentialField[fieldID * fieldWidth * fieldWidth + loop1 * fieldWidth + loop2] = countFieldTile(x, y, goalX[fieldID], goalY[fieldID]);
		}
	}
}